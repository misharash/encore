#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <complex>
#include "gpufuncs.h"
#include <thrust/complex.h>
#include <hip/hip_complex.h>

int count = 0;

__global__ void add_to_power4_kernel(double *fourpcf, double *weight4pcf,
	thrust::complex<double>* alm, thrust::complex<double> *almconj,
	int *lut4_l1, int *lut4_l2, int *lut4_l3, int *lut4_n,
	int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k, 
        double wp, int nb, int norder, int nlm, int nouter, int ninner) {
    //thread index i
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nouter * ninner) return;
    //compute indices for LUTs
    int iouter = i/ninner;
    int iinner = i%ninner;
    //calc bin_index
    int bin_index = lut4_zeta[iouter]+iinner;
    double pcf_element = fourpcf[bin_index]; // this element
    //cald weight
    double weight = weight4pcf[lut4_n[iouter]];
    //outer loop indices
    int l1 = lut4_l1[iouter];
    int tmp_l1 = l1*(l1+1)/2;
    int l2 = lut4_l2[iouter];
    int tmp_l2 = l2*(l2+1)/2;
    int l3 = lut4_l3[iouter];
    int tmp_l3 = l3*(l3+1)/2;
    int n = lut4_n[iouter]; //this is the starting n for this thread
    //inner loop indices
    int ii = lut4_i[iinner];
    int j = lut4_j[iinner];
    int k = lut4_k[iinner];
    //alms
    thrust::complex<double> alm1w = 0;
    thrust::complex<double> alm2 = 0;
    int m3, tmp_lm3;
    double delta;
    //now loop over ms on this thread
    // Iterate over all m1 (including negative)
    for(int m1=-l1; m1<=l1; m1++){
      // Create temporary copy of primary_weight*a_l1m1, taking conjugate if necessary [(-1)^m factor is absorbed into weight]
      if (m1 < 0) alm1w = wp*almconj[ii*nlm+tmp_l1-m1]; else alm1w = wp*alm[ii*nlm+tmp_l1+m1];
      // Iterate over all m2 (including negative)
      for(int m2=-l2; m2<=l2; m2++){
        m3 = -m1-m2;
        if (m3<0) continue; // only need to use m3>=0
        if (m3>l3) continue; // this violates triangle conditions

        // Look up the relevant weight
        weight = weight4pcf[n++];
        if (weight==0) continue;
        tmp_lm3 = tmp_l3+m3;
        // Create temporary copy of a_l2m2 and a_l3m3, taking conjugate if necessary
        // No conjugates needed for a_l3m3 since we fixed m3>=0!
        // Note we add the coupling weight factor to a_l3m3
        if (m2 < 0) alm2 = alm1w*almconj[j*nlm+tmp_l2-m2]; else alm2 = alm1w*alm[j*nlm+tmp_l2+m2];
        
	//calculate delta
        delta = weight*(alm2*alm[k*nlm+tmp_lm3]).real();
	//add to this element
	pcf_element += delta;
      }
    }
    fourpcf[bin_index] = pcf_element; //copy back to global memory 
}

__global__ void add_to_power4_kernel_float(float *fourpcf, float *weight4pcf,
        thrust::complex<float>* alm, thrust::complex<float> *almconj,
        int *lut4_l1, int *lut4_l2, int *lut4_l3, int *lut4_n,
        int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k,
        float wp, int nb, int norder, int nlm, int nouter, int ninner) {
    //thread index i
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nouter * ninner) return;
    //compute indices for LUTs
    int iouter = i/ninner;
    int iinner = i%ninner;
    //calc bin_index
    int bin_index = lut4_zeta[iouter]+iinner;
    float pcf_element = fourpcf[bin_index]; // this element
    //cald weight
    float weight = weight4pcf[lut4_n[iouter]];
    //outer loop indices
    int l1 = lut4_l1[iouter];
    int tmp_l1 = l1*(l1+1)/2;
    int l2 = lut4_l2[iouter];
    int tmp_l2 = l2*(l2+1)/2;
    int l3 = lut4_l3[iouter];
    int tmp_l3 = l3*(l3+1)/2;
    int n = lut4_n[iouter]; //this is the starting n for this thread
    //inner loop indices
    int ii = lut4_i[iinner];
    int j = lut4_j[iinner];
    int k = lut4_k[iinner];
    //alms
    thrust::complex<float> alm1w = 0;
    thrust::complex<float> alm2 = 0;
    int m3, tmp_lm3;
    float delta;
    //now loop over ms on this thread
    // Iterate over all m1 (including negative)
    for(int m1=-l1; m1<=l1; m1++){
      // Create temporary copy of primary_weight*a_l1m1, taking conjugate if necessary [(-1)^m factor is absorbed into weight]
      if (m1 < 0) alm1w = wp*almconj[ii*nlm+tmp_l1-m1]; else alm1w = wp*alm[ii*nlm+tmp_l1+m1];
      // Iterate over all m2 (including negative)
      for(int m2=-l2; m2<=l2; m2++){
        m3 = -m1-m2;
        if (m3<0) continue; // only need to use m3>=0
        if (m3>l3) continue; // this violates triangle conditions

        // Look up the relevant weight
        weight = weight4pcf[n++];
        if (weight==0) continue;
        tmp_lm3 = tmp_l3+m3;
        // Create temporary copy of a_l2m2 and a_l3m3, taking conjugate if necessary
        // No conjugates needed for a_l3m3 since we fixed m3>=0!
        // Note we add the coupling weight factor to a_l3m3
        if (m2 < 0) alm2 = alm1w*almconj[j*nlm+tmp_l2-m2]; else alm2 = alm1w*alm[j*nlm+tmp_l2+m2];

        //calculate delta
        delta = weight*(alm2*alm[k*nlm+tmp_lm3]).real();
        //add to this element
        pcf_element += delta;
      }
    }
    fourpcf[bin_index] = pcf_element; //copy back to global memory 
}

__global__ void add_to_power4_kernel_mixed(double *fourpcf, double *weight4pcf,
        thrust::complex<float>* alm, thrust::complex<float> *almconj,
        int *lut4_l1, int *lut4_l2, int *lut4_l3, int *lut4_n,
        int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k,
        float wp, int nb, int norder, int nlm, int nouter, int ninner) {
    //thread index i
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nouter * ninner) return;
    //compute indices for LUTs
    int iouter = i/ninner;
    int iinner = i%ninner;
    //calc bin_index
    int bin_index = lut4_zeta[iouter]+iinner;
    double pcf_element = fourpcf[bin_index]; // this element
    //cald weight
    double weight = weight4pcf[lut4_n[iouter]];
    //outer loop indices
    int l1 = lut4_l1[iouter];
    int tmp_l1 = l1*(l1+1)/2;
    int l2 = lut4_l2[iouter];
    int tmp_l2 = l2*(l2+1)/2;
    int l3 = lut4_l3[iouter];
    int tmp_l3 = l3*(l3+1)/2;
    int n = lut4_n[iouter]; //this is the starting n for this thread
    //inner loop indices
    int ii = lut4_i[iinner];
    int j = lut4_j[iinner];
    int k = lut4_k[iinner];
    //alms
    thrust::complex<float> alm1w = 0;
    thrust::complex<float> alm2 = 0;
    int m3, tmp_lm3;
    double delta;
    //now loop over ms on this thread
    // Iterate over all m1 (including negative)
    for(int m1=-l1; m1<=l1; m1++){
      // Create temporary copy of primary_weight*a_l1m1, taking conjugate if necessary [(-1)^m factor is absorbed into weight]
      if (m1 < 0) alm1w = wp*almconj[ii*nlm+tmp_l1-m1]; else alm1w = wp*alm[ii*nlm+tmp_l1+m1];
      // Iterate over all m2 (including negative)
      for(int m2=-l2; m2<=l2; m2++){
        m3 = -m1-m2;
        if (m3<0) continue; // only need to use m3>=0
        if (m3>l3) continue; // this violates triangle conditions

        // Look up the relevant weight
        weight = weight4pcf[n++];
        if (weight==0) continue;
        tmp_lm3 = tmp_l3+m3;
        // Create temporary copy of a_l2m2 and a_l3m3, taking conjugate if necessary
        // No conjugates needed for a_l3m3 since we fixed m3>=0!
        // Note we add the coupling weight factor to a_l3m3
        if (m2 < 0) alm2 = alm1w*almconj[j*nlm+tmp_l2-m2]; else alm2 = alm1w*alm[j*nlm+tmp_l2+m2];

        //calculate delta
        delta = weight*(alm2*alm[k*nlm+tmp_lm3]).real();
        //add to this element
        pcf_element += delta;
      }
    }
    fourpcf[bin_index] = pcf_element; //copy back to global memory 
}

__global__ void add_to_power4_kernel_orig(double *fourpcf, double *weight4pcf,
        thrust::complex<double>* alm, thrust::complex<double> *almconj,
        int *lut4_l1, int *lut4_l2, int *lut4_l3, int *lut4_m1, int *lut4_m2,
	int *lut4_n, int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k,
        double wp, int nb, int norder, int nlm, int nouter, int ninner) {
    //thread index i
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nouter * ninner) return;
    //compute indices for LUTs
    int iouter = i/ninner;
    int iinner = i%ninner;
    //calc bin_index
    int bin_index = lut4_zeta[iouter]+iinner;
    //outer loop indices
    int l1 = lut4_l1[iouter];
    int tmp_l1 = l1*(l1+1)/2;
    int l2 = lut4_l2[iouter];
    int tmp_l2 = l2*(l2+1)/2;
    int l3 = lut4_l3[iouter];
    int tmp_l3 = l3*(l3+1)/2;
    int m1 = lut4_m1[iouter];
    int m2 = lut4_m2[iouter];
    int n = lut4_n[iouter]; 
    //calc weight
    double weight = weight4pcf[n];
    //inner loop indices
    int ii = lut4_i[iinner];
    int j = lut4_j[iinner];
    int k = lut4_k[iinner];
    //alms
    thrust::complex<double> alm1w = 0;
    thrust::complex<double> alm2 = 0;
    if (m1 < 0) alm1w = wp*almconj[ii*nlm+tmp_l1-m1]; else alm1w = wp*alm[ii*nlm+tmp_l1+m1];
    if (m2 < 0) alm2 = alm1w*almconj[j*nlm+tmp_l2-m2]; else alm2 = alm1w*alm[j*nlm+tmp_l2+m2];
    int m3 = -m1-m2;
    int tmp_lm3 = tmp_l3+m3;
    double delta = weight*(alm2*alm[k*nlm+tmp_lm3]).real();
    atomicAdd(&fourpcf[bin_index], delta);
}

__global__ void add_to_power4_kernel_orig_float(float *fourpcf, float *weight4pcf,
        thrust::complex<float>* alm, thrust::complex<float> *almconj,
        int *lut4_l1, int *lut4_l2, int *lut4_l3, int *lut4_m1, int *lut4_m2,
        int *lut4_n, int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k,
        float wp, int nb, int norder, int nlm, int nouter, int ninner) {
    //thread index i
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nouter * ninner) return;
    //compute indices for LUTs
    int iouter = i/ninner;
    int iinner = i%ninner;
    //calc bin_index
    int bin_index = lut4_zeta[iouter]+iinner;
    //outer loop indices
    int l1 = lut4_l1[iouter];
    int tmp_l1 = l1*(l1+1)/2;
    int l2 = lut4_l2[iouter];
    int tmp_l2 = l2*(l2+1)/2;
    int l3 = lut4_l3[iouter];
    int tmp_l3 = l3*(l3+1)/2;
    int m1 = lut4_m1[iouter];
    int m2 = lut4_m2[iouter];
    int n = lut4_n[iouter]; 
    //calc weight
    float weight = weight4pcf[n];
    //inner loop indices
    int ii = lut4_i[iinner];
    int j = lut4_j[iinner];
    int k = lut4_k[iinner];
    //alms
    thrust::complex<float> alm1w = 0;
    thrust::complex<float> alm2 = 0;
    if (m1 < 0) alm1w = wp*almconj[ii*nlm+tmp_l1-m1]; else alm1w = wp*alm[ii*nlm+tmp_l1+m1];
    if (m2 < 0) alm2 = alm1w*almconj[j*nlm+tmp_l2-m2]; else alm2 = alm1w*alm[j*nlm+tmp_l2+m2];
    int m3 = -m1-m2;
    int tmp_lm3 = tmp_l3+m3;
    float delta = weight*(alm2*alm[k*nlm+tmp_lm3]).real();
    atomicAdd(&fourpcf[bin_index], delta);
}

__global__ void add_to_power4_kernel_orig_mixed(double *fourpcf, double *weight4pcf,
        thrust::complex<float>* alm, thrust::complex<float> *almconj,
        int *lut4_l1, int *lut4_l2, int *lut4_l3, int *lut4_m1, int *lut4_m2,
        int *lut4_n, int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k,
        float wp, int nb, int norder, int nlm, int nouter, int ninner) {
    //thread index i
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nouter * ninner) return;
    //compute indices for LUTs
    int iouter = i/ninner;
    int iinner = i%ninner;
    //calc bin_index
    int bin_index = lut4_zeta[iouter]+iinner;
    //outer loop indices
    int l1 = lut4_l1[iouter];
    int tmp_l1 = l1*(l1+1)/2;
    int l2 = lut4_l2[iouter];
    int tmp_l2 = l2*(l2+1)/2;
    int l3 = lut4_l3[iouter];
    int tmp_l3 = l3*(l3+1)/2;
    int m1 = lut4_m1[iouter];
    int m2 = lut4_m2[iouter];
    int n = lut4_n[iouter]; 
    //calc weight
    double weight = weight4pcf[n];
    //inner loop indices
    int ii = lut4_i[iinner];
    int j = lut4_j[iinner];
    int k = lut4_k[iinner];
    //alms
    thrust::complex<float> alm1w = 0;
    thrust::complex<float> alm2 = 0;
    if (m1 < 0) alm1w = wp*almconj[ii*nlm+tmp_l1-m1]; else alm1w = wp*alm[ii*nlm+tmp_l1+m1];
    if (m2 < 0) alm2 = alm1w*almconj[j*nlm+tmp_l2-m2]; else alm2 = alm1w*alm[j*nlm+tmp_l2+m2];
    int m3 = -m1-m2;
    int tmp_lm3 = tmp_l3+m3;
    double delta = weight*(alm2*alm[k*nlm+tmp_lm3]).real();
    atomicAdd(&fourpcf[bin_index], delta);
}

__global__ void add_to_power5_kernel(double *fivepcf, double *weight5pcf, thrust::complex<double>* alm,
	thrust::complex<double> *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l12, int *lut5_l3,
        int *lut5_l4, int *lut5_n, int *lut5_zeta, int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
        double wp, int nb, int norder, int nlm, int nouter, int ninner) {
    //thread index i
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nouter * ninner) return;
    //compute indices for LUTs
    int iouter = i/ninner;
    int iinner = i%ninner;
    //calc bin_index
    int bin_index = lut5_zeta[iouter]+iinner;
    double pcf_element = fivepcf[bin_index]; // this element
    //cald weight
    double weight = weight5pcf[lut5_n[iouter]];
    //outer loop indices
    int l1 = lut5_l1[iouter];
    int tmp_l1 = l1*(l1+1)/2;
    int l2 = lut5_l2[iouter];
    int tmp_l2 = l2*(l2+1)/2;
    int l12 = lut5_l12[iouter];
    int l3 = lut5_l3[iouter];
    int tmp_l3 = l3*(l3+1)/2;
    int l4 = lut5_l4[iouter];
    int tmp_l4 = l4*(l4+1)/2;
    int n = lut5_n[iouter]; //this is the starting n for this thread
    //inner loop indices
    int ii = lut5_i[iinner];
    int j = lut5_j[iinner];
    int k = lut5_k[iinner];
    int l = lut5_l[iinner];
    //alms
    thrust::complex<double> alm1w = 0;
    thrust::complex<double> alm2 = 0;
    thrust::complex<double> alm3 = 0;
    int m4, tmp_lm4;
    double delta;
    //now loop over ms on this thread
    // Iterate over all m1 (including negative)
    for(int m1=-l1; m1<=l1; m1++){
      // Create temporary copy of primary_weight*a_l1m1, taking conjugate if necessary [(-1)^m factor is absorbed into weight]
      if (m1 < 0) alm1w = wp*almconj[ii*nlm+tmp_l1-m1]; else alm1w = wp*alm[ii*nlm+tmp_l1+m1];
      // Iterate over all m2 (including negative)
      for(int m2=-l2; m2<=l2; m2++){
        if(abs(m1+m2)>l12) continue; // m12 condition
        // Create temporary copy of a_l2m2, taking conjugate if necessary
        if (m2 < 0) alm2 = alm1w*almconj[j*nlm+tmp_l2-m2]; else alm2 = alm1w*alm[j*nlm+tmp_l2+m2];
        // Iterate over m3 (including negative)
        for(int m3=-l3; m3<=l3; m3++){
          m4 = -m1-m2-m3;
          if (m4<0) continue; // only need to use m4>=0
          if (m4>l4) continue; // this violates triangle conditions
          // Look up the relevant weight
          weight = weight5pcf[n++];
          if (weight==0) continue;
          tmp_lm4 = tmp_l4+m4;
          // Create temporary copies of a_l3m3 and a_l4m4, taking conjugates if necessary
          // No conjugates needed for a_l4m4 since we fixed m4>=0!
          // Note we add the coupling weight factor to a_l4m4
          if (m3 < 0) alm3 = alm2*almconj[k*nlm+tmp_l3-m3]; else alm3 = alm2*alm[k*nlm+tmp_l3+m3];
	  //calculate delta
          delta = weight*(alm3*alm[l*nlm+tmp_lm4]).real();
	  //add to this element
	  pcf_element += delta;
        }
      }
    }
    fivepcf[bin_index] = pcf_element; //copy back to global memory 
}

__global__ void add_to_power5_kernel_float(float *fivepcf, float *weight5pcf, thrust::complex<float>* alm,
        thrust::complex<float> *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l12, int *lut5_l3,
        int *lut5_l4, int *lut5_n, int *lut5_zeta, int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
	float wp, int nlm, int nouter, int ninner) {
    //thread index i
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nouter * ninner) return;
    //compute indices for LUTs
    int iouter = i/ninner;
    int iinner = i%ninner;
    //calc bin_index
    int bin_index = lut5_zeta[iouter]+iinner;
    float pcf_element = fivepcf[bin_index]; // this element
    //calc weight
    float weight = weight5pcf[lut5_n[iouter]];
    //outer loop indices
    int l1 = lut5_l1[iouter];
    int tmp_l1 = l1*(l1+1)/2;
    int l2 = lut5_l2[iouter];
    int tmp_l2 = l2*(l2+1)/2;
    int l12 = lut5_l12[iouter];
    int l3 = lut5_l3[iouter];
    int tmp_l3 = l3*(l3+1)/2;
    int l4 = lut5_l4[iouter];
    int tmp_l4 = l4*(l4+1)/2;
    int n = lut5_n[iouter]; //this is the starting n for this thread
    //inner loop indices
    int ii = lut5_i[iinner];
    int j = lut5_j[iinner];
    int k = lut5_k[iinner];
    int l = lut5_l[iinner];
    //alms
    thrust::complex<float> alm1w = 0;
    thrust::complex<float> alm2 = 0;
    thrust::complex<float> alm3 = 0;
    int m4, tmp_lm4;
    float delta;
    //now loop over ms on this thread
    // Iterate over all m1 (including negative)
    for(int m1=-l1; m1<=l1; m1++){
      // Create temporary copy of primary_weight*a_l1m1, taking conjugate if necessary [(-1)^m factor is absorbed into weight]
      if (m1 < 0) alm1w = wp*almconj[ii*nlm+tmp_l1-m1]; else alm1w = wp*alm[ii*nlm+tmp_l1+m1];
      // Iterate over all m2 (including negative)
      for(int m2=-l2; m2<=l2; m2++){
        if(abs(m1+m2)>l12) continue; // m12 condition
        // Create temporary copy of a_l2m2, taking conjugate if necessary
        if (m2 < 0) alm2 = alm1w*almconj[j*nlm+tmp_l2-m2]; else alm2 = alm1w*alm[j*nlm+tmp_l2+m2];
        // Iterate over m3 (including negative)
        for(int m3=-l3; m3<=l3; m3++){
          m4 = -m1-m2-m3;
          if (m4<0) continue; // only need to use m4>=0
          if (m4>l4) continue; // this violates triangle conditions
          // Look up the relevant weight
          weight = weight5pcf[n++];
          if (weight==0) continue;
          tmp_lm4 = tmp_l4+m4;
          // Create temporary copies of a_l3m3 and a_l4m4, taking conjugates if necessary
          // No conjugates needed for a_l4m4 since we fixed m4>=0!
          // Note we add the coupling weight factor to a_l4m4
          if (m3 < 0) alm3 = alm2*almconj[k*nlm+tmp_l3-m3]; else alm3 = alm2*alm[k*nlm+tmp_l3+m3];
          //calculate delta
          delta = weight*(alm3*alm[l*nlm+tmp_lm4]).real();
          //add to this element
          pcf_element += delta;
        }
      }
    }
    fivepcf[bin_index] = pcf_element; 
}

__global__ void add_to_power5_kernel_mixed(double *fivepcf, double *weight5pcf, thrust::complex<float>* alm,
        thrust::complex<float> *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l12, int *lut5_l3,
        int *lut5_l4, int *lut5_n, int *lut5_zeta, int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
	float wp, int nlm, int nouter, int ninner) {
    //thread index i
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nouter * ninner) return;
    //compute indices for LUTs
    int iouter = i/ninner;
    int iinner = i%ninner;
    //calc bin_index
    int bin_index = lut5_zeta[iouter]+iinner;
    double pcf_element = fivepcf[bin_index]; // this element
    //calc weight
    double weight = weight5pcf[lut5_n[iouter]];
    //outer loop indices
    int l1 = lut5_l1[iouter];
    int tmp_l1 = l1*(l1+1)/2;
    int l2 = lut5_l2[iouter];
    int tmp_l2 = l2*(l2+1)/2;
    int l12 = lut5_l12[iouter];
    int l3 = lut5_l3[iouter];
    int tmp_l3 = l3*(l3+1)/2;
    int l4 = lut5_l4[iouter];
    int tmp_l4 = l4*(l4+1)/2;
    int n = lut5_n[iouter]; //this is the starting n for this thread
    //inner loop indices
    int ii = lut5_i[iinner];
    int j = lut5_j[iinner];
    int k = lut5_k[iinner];
    int l = lut5_l[iinner];
    //alms
    thrust::complex<float> alm1w = 0;
    thrust::complex<float> alm2 = 0;
    thrust::complex<float> alm3 = 0;
    int m4, tmp_lm4;
    double delta;
    //now loop over ms on this thread
    // Iterate over all m1 (including negative)
    for(int m1=-l1; m1<=l1; m1++){
      // Create temporary copy of primary_weight*a_l1m1, taking conjugate if necessary [(-1)^m factor is absorbed into weight]
      if (m1 < 0) alm1w = wp*almconj[ii*nlm+tmp_l1-m1]; else alm1w = wp*alm[ii*nlm+tmp_l1+m1];
      // Iterate over all m2 (including negative)
      for(int m2=-l2; m2<=l2; m2++){
        if(abs(m1+m2)>l12) continue; // m12 condition
        // Create temporary copy of a_l2m2, taking conjugate if necessary
        if (m2 < 0) alm2 = alm1w*almconj[j*nlm+tmp_l2-m2]; else alm2 = alm1w*alm[j*nlm+tmp_l2+m2];
        // Iterate over m3 (including negative)
        for(int m3=-l3; m3<=l3; m3++){
          m4 = -m1-m2-m3;
          if (m4<0) continue; // only need to use m4>=0
          if (m4>l4) continue; // this violates triangle conditions
          // Look up the relevant weight
          weight = weight5pcf[n++];
          if (weight==0) continue;
          tmp_lm4 = tmp_l4+m4;
          // Create temporary copies of a_l3m3 and a_l4m4, taking conjugates if necessary
          // No conjugates needed for a_l4m4 since we fixed m4>=0!
          // Note we add the coupling weight factor to a_l4m4
          if (m3 < 0) alm3 = alm2*almconj[k*nlm+tmp_l3-m3]; else alm3 = alm2*alm[k*nlm+tmp_l3+m3];
          //calculate delta
          delta = weight*(alm3*alm[l*nlm+tmp_lm4]).real();
          //add to this element
          pcf_element += delta;
        }
      }
    }
    fivepcf[bin_index] = pcf_element; 
}

__global__ void add_to_power5_kernel_orig(double *fivepcf, double *weight5pcf, thrust::complex<double>* alm,
	thrust::complex<double> *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l3,
        int *lut5_l4, int *lut5_m1, int *lut5_m2, int *lut5_m3, int *lut5_n, int *lut5_zeta,
	int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
        double wp, int nb, int norder, int nlm, int nouter, int ninner) {
    //thread index i
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nouter * ninner) return;
    //compute indices for LUTs
    int iouter = i/ninner;
    int iinner = i%ninner;
    //calc bin_index
    int bin_index = lut5_zeta[iouter]+iinner;
    //outer loop indices
    int l1 = lut5_l1[iouter];
    int tmp_l1 = l1*(l1+1)/2;
    int l2 = lut5_l2[iouter];
    int tmp_l2 = l2*(l2+1)/2;
    int l3 = lut5_l3[iouter];
    int tmp_l3 = l3*(l3+1)/2;
    int l4 = lut5_l4[iouter];
    int tmp_l4 = l4*(l4+1)/2;
    int m1 = lut5_m1[iouter];
    int m2 = lut5_m2[iouter];
    int m3 = lut5_m3[iouter];
    int n = lut5_n[iouter]; //this is the starting n for this thread
    //calc weight
    double weight = weight5pcf[n];
    //inner loop indices
    int ii = lut5_i[iinner];
    int j = lut5_j[iinner];
    int k = lut5_k[iinner];
    int l = lut5_l[iinner];
    //alms
    thrust::complex<double> alm1w = 0;
    thrust::complex<double> alm2 = 0;
    thrust::complex<double> alm3 = 0;
    if (m1 < 0) alm1w = wp*almconj[ii*nlm+tmp_l1-m1]; else alm1w = wp*alm[ii*nlm+tmp_l1+m1];
    if (m2 < 0) alm2 = alm1w*almconj[j*nlm+tmp_l2-m2]; else alm2 = alm1w*alm[j*nlm+tmp_l2+m2];
    int m4 = -m1-m2-m3;
    int tmp_lm4 = tmp_l4+m4;
    if (m3 < 0) alm3 = alm2*almconj[k*nlm+tmp_l3-m3]; else alm3 = alm2*alm[k*nlm+tmp_l3+m3];
    double delta = weight*(alm3*alm[l*nlm+tmp_lm4]).real();
    atomicAdd(&fivepcf[bin_index], delta);
    //atomicAdd(&fivepcf[bin_index], m2);
}

__global__ void add_to_power5_kernel_orig_float(float *fivepcf, float *weight5pcf, thrust::complex<float>* alm,
	thrust::complex<float> *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l3,
        int *lut5_l4, int *lut5_m1, int *lut5_m2, int *lut5_m3, int *lut5_n, int *lut5_zeta,
	int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
        float wp, int nb, int norder, int nlm, int nouter, int ninner) {
    //thread index i
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nouter * ninner) return;
    //compute indices for LUTs
    int iouter = i/ninner;
    int iinner = i%ninner;
    //calc bin_index
    int bin_index = lut5_zeta[iouter]+iinner;
    //outer loop indices
    int l1 = lut5_l1[iouter];
    int tmp_l1 = l1*(l1+1)/2;
    int l2 = lut5_l2[iouter];
    int tmp_l2 = l2*(l2+1)/2;
    int l3 = lut5_l3[iouter];
    int tmp_l3 = l3*(l3+1)/2;
    int l4 = lut5_l4[iouter];
    int tmp_l4 = l4*(l4+1)/2;
    int m1 = lut5_m1[iouter];
    int m2 = lut5_m2[iouter];
    int m3 = lut5_m3[iouter];
    int n = lut5_n[iouter]; //this is the starting n for this thread
    //calc weight
    float weight = weight5pcf[n];
    //inner loop indices
    int ii = lut5_i[iinner];
    int j = lut5_j[iinner];
    int k = lut5_k[iinner];
    int l = lut5_l[iinner];
    //alms
    thrust::complex<float> alm1w = 0;
    thrust::complex<float> alm2 = 0;
    thrust::complex<float> alm3 = 0;
    if (m1 < 0) alm1w = wp*almconj[ii*nlm+tmp_l1-m1]; else alm1w = wp*alm[ii*nlm+tmp_l1+m1];
    if (m2 < 0) alm2 = alm1w*almconj[j*nlm+tmp_l2-m2]; else alm2 = alm1w*alm[j*nlm+tmp_l2+m2];
    int m4 = -m1-m2-m3;
    int tmp_lm4 = tmp_l4+m4;
    if (m3 < 0) alm3 = alm2*almconj[k*nlm+tmp_l3-m3]; else alm3 = alm2*alm[k*nlm+tmp_l3+m3];
    float delta = weight*(alm3*alm[l*nlm+tmp_lm4]).real();
    atomicAdd(&fivepcf[bin_index], delta);
}

__global__ void add_to_power5_kernel_orig_mixed(double *fivepcf, double *weight5pcf, thrust::complex<float>* alm,
	thrust::complex<float> *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l3,
        int *lut5_l4, int *lut5_m1, int *lut5_m2, int *lut5_m3, int *lut5_n, int *lut5_zeta,
	int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
        float wp, int nb, int norder, int nlm, int nouter, int ninner) {
    //thread index i
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= nouter * ninner) return;
    //compute indices for LUTs
    int iouter = i/ninner;
    int iinner = i%ninner;
    //calc bin_index
    int bin_index = lut5_zeta[iouter]+iinner;
    //outer loop indices
    int l1 = lut5_l1[iouter];
    int tmp_l1 = l1*(l1+1)/2;
    int l2 = lut5_l2[iouter];
    int tmp_l2 = l2*(l2+1)/2;
    int l3 = lut5_l3[iouter];
    int tmp_l3 = l3*(l3+1)/2;
    int l4 = lut5_l4[iouter];
    int tmp_l4 = l4*(l4+1)/2;
    int m1 = lut5_m1[iouter];
    int m2 = lut5_m2[iouter];
    int m3 = lut5_m3[iouter];
    int n = lut5_n[iouter]; //this is the starting n for this thread
    //calc weight
    double weight = weight5pcf[n];
    //inner loop indices
    int ii = lut5_i[iinner];
    int j = lut5_j[iinner];
    int k = lut5_k[iinner];
    int l = lut5_l[iinner];
    //alms
    thrust::complex<float> alm1w = 0;
    thrust::complex<float> alm2 = 0;
    thrust::complex<float> alm3 = 0;
    if (m1 < 0) alm1w = wp*almconj[ii*nlm+tmp_l1-m1]; else alm1w = wp*alm[ii*nlm+tmp_l1+m1];
    if (m2 < 0) alm2 = alm1w*almconj[j*nlm+tmp_l2-m2]; else alm2 = alm1w*alm[j*nlm+tmp_l2+m2];
    int m4 = -m1-m2-m3;
    int tmp_lm4 = tmp_l4+m4;
    if (m3 < 0) alm3 = alm2*almconj[k*nlm+tmp_l3-m3]; else alm3 = alm2*alm[k*nlm+tmp_l3+m3];
    double delta = weight*(alm3*alm[l*nlm+tmp_lm4]).real();
    atomicAdd(&fivepcf[bin_index], delta);
}

void gpu_allocate_luts4(int **p_lut4_l1, int **p_lut4_l2, int **p_lut4_l3, int **p_lut4_n,
	int **p_lut4_zeta, int **p_lut4_i, int **p_lut4_j, int **p_lut4_k,
        int nouter, int ninner) {
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&(*p_lut4_l1), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut4_l2), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut4_l3), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut4_n), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut4_zeta), nouter*sizeof(int));

  hipMallocManaged(&(*p_lut4_i), ninner*sizeof(int));
  hipMallocManaged(&(*p_lut4_j), ninner*sizeof(int));
  hipMallocManaged(&(*p_lut4_k), ninner*sizeof(int));
}

void gpu_allocate_m_luts4(int **p_lut4_m1, int **p_lut4_m2, int nouter) {
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&(*p_lut4_m1), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut4_m2), nouter*sizeof(int));
}

void gpu_allocate_fourpcf(double **p_fourpcf, double *fourpcf, int size) {
  //hipMalloc(&(*p_fourpcf), size*sizeof(double));
  //hipMemcpy((*p_fourpcf), fourpcf, size, hipMemcpyHostToDevice);
  //use MallocManaged because of weirdness with hipMemcpy not seeming to work with weight4pcf
  hipMallocManaged(&(*p_fourpcf), size*sizeof(double));
  double *d_fourpcf = *(p_fourpcf);
  for (int i = 0; i < size; i++) d_fourpcf[i] = fourpcf[i];
}

void gpu_allocate_weight4pcf(double **p_weight4pcf, double *weight4pcf, int size) {
  //hipMalloc(&(*p_weight4pcf), size*sizeof(double));
  //hipMemcpy((*p_weight4pcf), weight4pcf, size, hipMemcpyHostToDevice);
  //use MallocManaged because of weirdness with hipMemcpy not seeming to work with weight4pcf
  hipMallocManaged(&(*p_weight4pcf), size*sizeof(double));
  double *d_weight4pcf = *(p_weight4pcf);
  for (int i = 0; i < size; i++) d_weight4pcf[i] = weight4pcf[i];
}

void copy_fourpcf(double **p_fourpcf, double *fourpcf, int size) {
  hipMemcpy(fourpcf, (*p_fourpcf), size*sizeof(double), hipMemcpyDeviceToHost);
}

void gpu_allocate_fourpcf(float **p_fourpcf, double *fourpcf, int size) {
  hipMallocManaged(&(*p_fourpcf), size*sizeof(float));
  float *f_fourpcf = *(p_fourpcf);
  for (int i = 0; i < size; i++) f_fourpcf[i] = (float)fourpcf[i];
}

void gpu_allocate_weight4pcf(float **p_weight4pcf, double *weight4pcf, int size) {
  hipMallocManaged(&(*p_weight4pcf), size*sizeof(float));
  float *f_weight4pcf = *(p_weight4pcf);
  for (int i = 0; i < size; i++) f_weight4pcf[i] = (float)weight4pcf[i];
}

void copy_fourpcf(float **p_fourpcf, double *fourpcf, int size) {
  float *f_fourpcf = *(p_fourpcf);
  for (int i = 0; i < size; i++) fourpcf[i] = (double)f_fourpcf[i];
}

void gpu_free_luts4(int *lut4_l1, int *lut4_l2, int *lut4_l3, int *lut4_n,
	int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k) {
  hipFree(lut4_l1);
  hipFree(lut4_l2);
  hipFree(lut4_l3);
  hipFree(lut4_n);
  hipFree(lut4_zeta);
  hipFree(lut4_i);
  hipFree(lut4_j);
  hipFree(lut4_k);
}

void gpu_free_memory4(double *fourpcf, double *weight4pcf) {
  hipFree(fourpcf);
  hipFree(weight4pcf);
}

void gpu_free_memory4(float *fourpcf, float *weight4pcf) {
  hipFree(fourpcf);
  hipFree(weight4pcf);
}

void gpu_free_memory_m4(int *lut4_m1, int *lut4_m2) {
  hipFree(lut4_m1);
  hipFree(lut4_m2);
}

void gpu_allocate_luts(int **p_lut5_l1, int **p_lut5_l2, int **p_lut5_l12, int **p_lut5_l3,
        int **p_lut5_l4, int **p_lut5_n,
        int **p_lut5_zeta, int **p_lut5_i, int **p_lut5_j, int **p_lut5_k, int **p_lut5_l,
        int nouter, int ninner) {
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&(*p_lut5_l1), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut5_l2), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut5_l12), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut5_l3), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut5_l4), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut5_n), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut5_zeta), nouter*sizeof(int));

  hipMallocManaged(&(*p_lut5_i), ninner*sizeof(int));
  hipMallocManaged(&(*p_lut5_j), ninner*sizeof(int));
  hipMallocManaged(&(*p_lut5_k), ninner*sizeof(int));
  hipMallocManaged(&(*p_lut5_l), ninner*sizeof(int));
}

void gpu_allocate_m_luts(int **p_lut5_m1, int **p_lut5_m2, int **p_lut5_m3, int nouter) {
  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&(*p_lut5_m1), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut5_m2), nouter*sizeof(int));
  hipMallocManaged(&(*p_lut5_m3), nouter*sizeof(int));
}

void gpu_allocate_fivepcf(double **p_fivepcf, double *fivepcf, int size) {
  //*p_fivepcf = (double *)malloc(sizeof(double)*size);
  //hipMalloc(&(*p_fivepcf), size*sizeof(double));
  //hipMemcpy((*p_fivepcf), fivepcf, size, hipMemcpyHostToDevice);
  //use MallocManaged because of weirdness with hipMemcpy not seeming to work with weight4pcf
  hipMallocManaged(&(*p_fivepcf), size*sizeof(double));
  double *d_fivepcf = *(p_fivepcf);
  for (int i = 0; i < size; i++) d_fivepcf[i] = fivepcf[i];
}

void gpu_allocate_weight5pcf(double **p_weight5pcf, double *weight5pcf, int size) {
  //hipMalloc(&(*p_weight5pcf), size*sizeof(double));
  //hipMemcpy((*p_weight5pcf), weight5pcf, size, hipMemcpyHostToDevice);
  //use MallocManaged because of weirdness with hipMemcpy not seeming to work with weight4pcf
  hipMallocManaged(&(*p_weight5pcf), size*sizeof(double));
  double *d_weight5pcf = *(p_weight5pcf);
  for (int i = 0; i < size; i++) d_weight5pcf[i] = weight5pcf[i];
}

void copy_fivepcf(double **p_fivepcf, double *fivepcf, int size) {
  hipMemcpy(fivepcf, (*p_fivepcf), size*sizeof(double), hipMemcpyDeviceToHost);
}

void gpu_allocate_fivepcf(float **p_fivepcf, double *fivepcf, int size) {
  hipMallocManaged(&(*p_fivepcf), size*sizeof(float));
  float *f_fivepcf = *(p_fivepcf);
  for (int i = 0; i < size; i++) f_fivepcf[i] = (float)fivepcf[i];
}

void gpu_allocate_weight5pcf(float **p_weight5pcf, double *weight5pcf, int size) {
  hipMallocManaged(&(*p_weight5pcf), size*sizeof(float));
  float *f_weight5pcf = *(p_weight5pcf);
  for (int i = 0; i < size; i++) f_weight5pcf[i] = (float)weight5pcf[i];
}

void copy_fivepcf(float **p_fivepcf, double *fivepcf, int size) {
  float *f_fivepcf = *(p_fivepcf);
  for (int i = 0; i < size; i++) fivepcf[i] = (double)f_fivepcf[i];
}

void gpu_free_luts(int *lut5_l1, int *lut5_l2, int *lut5_l12, int *lut5_l3,
        int *lut5_l4, int *lut5_n, int *lut5_zeta, int *lut5_i,
	int *lut5_j, int *lut5_k, int *lut5_l) {
  hipFree(lut5_l1);
  hipFree(lut5_l2);
  hipFree(lut5_l12);
  hipFree(lut5_l3);
  hipFree(lut5_l4);
  hipFree(lut5_n);
  hipFree(lut5_zeta);
  hipFree(lut5_i);
  hipFree(lut5_j);
  hipFree(lut5_k);
  hipFree(lut5_l);
}

void gpu_free_memory(double *fivepcf, double *weight5pcf) {
  hipFree(fivepcf);
  hipFree(weight5pcf);
}

void gpu_free_memory(float *fivepcf, float *weight5pcf) {
  hipFree(fivepcf);
  hipFree(weight5pcf);
}

void gpu_free_memory_m(int *lut5_m1, int *lut5_m2, int *lut5_m3) {
  hipFree(lut5_m1);
  hipFree(lut5_m2);
  hipFree(lut5_m3);
}

void gpu_add_to_power4(double *d_fourpcf, double *d_weight4pcf, Complex* alm,
        Complex *almconj, int *lut4_l1, int *lut4_l2, int *lut4_l3,
        int *lut4_n, int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k,
        double wp, int nb, int norder, int nlm, int nouter, int ninner, int nell4) {

  thrust::complex<double>* d_alm, *d_almconj;
  hipMalloc(&d_alm, nb*nlm*sizeof(thrust::complex<double>));
  hipMalloc(&d_almconj, nb*nlm*sizeof(thrust::complex<double>));

  hipMemcpy(d_alm, alm, nb*nlm*sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
  hipMemcpy(d_almconj, almconj, nb*nlm*sizeof(thrust::complex<double>), hipMemcpyHostToDevice);

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

if (count == 0) {
count++;
std::cout << "Threads = " << threads << " Nouter = " << nouter << " Ninner = " << ninner << std::endl;
}

  add_to_power4_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_fourpcf,
        d_weight4pcf, d_alm, d_almconj, lut4_l1, lut4_l2,
        lut4_l3, lut4_n, lut4_zeta, lut4_i, lut4_j, lut4_k, 
        wp, nb, norder, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  hipFree(d_alm);
  hipFree(d_almconj);
}

//float version of main kernel
void gpu_add_to_power4_float(float *d_fourpcf, float *d_weight4pcf, Complex* alm,
        Complex *almconj, int *lut4_l1, int *lut4_l2, int *lut4_l3,
        int *lut4_n, int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k,
        float wp, int nb, int norder, int nlm, int nouter, int ninner, int nell4) {

  thrust::complex<float>* d_alm, *d_almconj;
  hipMallocManaged(&d_alm, nb*nlm*sizeof(thrust::complex<float>));
  hipMallocManaged(&d_almconj, nb*nlm*sizeof(thrust::complex<float>));

  for (int i = 0; i < nb*nlm; i++) {
    d_alm[i] = (thrust::complex<float>)(alm[i]);
    d_almconj[i] = (thrust::complex<float>)(almconj[i]);
  }

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

  add_to_power4_kernel_float<<<blocksPerGrid, threadsPerBlock>>>(d_fourpcf,
        d_weight4pcf, d_alm, d_almconj, lut4_l1, lut4_l2,
        lut4_l3, lut4_n, lut4_zeta, lut4_i, lut4_j, lut4_k,
        wp, nb, norder, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  hipFree(d_alm);
  hipFree(d_almconj);
}

//mixed precision
void gpu_add_to_power4_mixed(double *d_fourpcf, double *d_weight4pcf, Complex* alm,
        Complex *almconj, int *lut4_l1, int *lut4_l2, int *lut4_l3,
        int *lut4_n, int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k,
        float wp, int nb, int norder, int nlm, int nouter, int ninner, int nell4) {

  thrust::complex<float>* d_alm, *d_almconj;
  hipMallocManaged(&d_alm, nb*nlm*sizeof(thrust::complex<float>));
  hipMallocManaged(&d_almconj, nb*nlm*sizeof(thrust::complex<float>));

  for (int i = 0; i < nb*nlm; i++) {
    d_alm[i] = (thrust::complex<float>)(alm[i]);
    d_almconj[i] = (thrust::complex<float>)(almconj[i]);
  }

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

  add_to_power4_kernel_mixed<<<blocksPerGrid, threadsPerBlock>>>(d_fourpcf,
        d_weight4pcf, d_alm, d_almconj, lut4_l1, lut4_l2,
        lut4_l3, lut4_n, lut4_zeta, lut4_i, lut4_j, lut4_k,
        wp, nb, norder, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  hipFree(d_alm);
  hipFree(d_almconj);
}

//alternate (original) kernel
void gpu_add_to_power4_orig(double *d_fourpcf, double *d_weight4pcf, Complex* alm,
        Complex *almconj, int *lut4_l1, int *lut4_l2, int *lut4_l3,
        int *lut4_m1, int *lut4_m2,
        int *lut4_n, int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k,
        double wp, int nb, int norder, int nlm, int nouter, int ninner, int nell4) {
  thrust::complex<double>* d_alm, *d_almconj;
  hipMalloc(&d_alm, nb*nlm*sizeof(thrust::complex<double>));
  hipMalloc(&d_almconj, nb*nlm*sizeof(thrust::complex<double>));

  hipMemcpy(d_alm, alm, nb*nlm*sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
  hipMemcpy(d_almconj, almconj, nb*nlm*sizeof(thrust::complex<double>), hipMemcpyHostToDevice);

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

if (count == 0) {
count++;
std::cout << "Threads = " << threads << std::endl;
}

  add_to_power4_kernel_orig<<<blocksPerGrid, threadsPerBlock>>>(d_fourpcf,
        d_weight4pcf, d_alm, d_almconj, lut4_l1, lut4_l2, lut4_l3,
	lut4_m1, lut4_m2, lut4_n, lut4_zeta, lut4_i, lut4_j, lut4_k,
        wp, nb, norder, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  hipFree(d_alm);
  hipFree(d_almconj);
}

//float version
void gpu_add_to_power4_orig_float(float *d_fourpcf, float *d_weight4pcf, Complex* alm,
        Complex *almconj, int *lut4_l1, int *lut4_l2, int *lut4_l3,
        int *lut4_m1, int *lut4_m2,
        int *lut4_n, int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k,
        float wp, int nb, int norder, int nlm, int nouter, int ninner, int nell4) {
  thrust::complex<float>* d_alm, *d_almconj;
  hipMallocManaged(&d_alm, nb*nlm*sizeof(thrust::complex<float>));
  hipMallocManaged(&d_almconj, nb*nlm*sizeof(thrust::complex<float>));

  for (int i = 0; i < nb*nlm; i++) {
    d_alm[i] = (thrust::complex<float>)(alm[i]);
    d_almconj[i] = (thrust::complex<float>)(almconj[i]);
  }

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

  add_to_power4_kernel_orig_float<<<blocksPerGrid, threadsPerBlock>>>(d_fourpcf,
        d_weight4pcf, d_alm, d_almconj, lut4_l1, lut4_l2, lut4_l3,
	lut4_m1, lut4_m2, lut4_n, lut4_zeta, lut4_i, lut4_j, lut4_k,
        wp, nb, norder, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  hipFree(d_alm);
  hipFree(d_almconj);
}

//mixed precision
void gpu_add_to_power4_orig_mixed(double *d_fourpcf, double *d_weight4pcf, Complex* alm,
        Complex *almconj, int *lut4_l1, int *lut4_l2, int *lut4_l3,
        int *lut4_m1, int *lut4_m2,
        int *lut4_n, int *lut4_zeta, int *lut4_i, int *lut4_j, int *lut4_k,
        float wp, int nb, int norder, int nlm, int nouter, int ninner, int nell4) {
  thrust::complex<float>* d_alm, *d_almconj;
  hipMallocManaged(&d_alm, nb*nlm*sizeof(thrust::complex<float>));
  hipMallocManaged(&d_almconj, nb*nlm*sizeof(thrust::complex<float>));

  for (int i = 0; i < nb*nlm; i++) {
    d_alm[i] = (thrust::complex<float>)(alm[i]);
    d_almconj[i] = (thrust::complex<float>)(almconj[i]);
  }

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

  add_to_power4_kernel_orig_mixed<<<blocksPerGrid, threadsPerBlock>>>(d_fourpcf,
        d_weight4pcf, d_alm, d_almconj, lut4_l1, lut4_l2, lut4_l3,
	lut4_m1, lut4_m2, lut4_n, lut4_zeta, lut4_i, lut4_j, lut4_k,
        wp, nb, norder, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  hipFree(d_alm);
  hipFree(d_almconj);
}

void gpu_add_to_power5(double *d_fivepcf, double *d_weight5pcf, Complex* alm,
        Complex *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l12, int *lut5_l3,
        int *lut5_l4, int *lut5_n,
        int *lut5_zeta, int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
	double wp, int nb, int norder, int nlm, int nouter, int ninner, int nell5) {

  thrust::complex<double>* d_alm, *d_almconj;
  hipMalloc(&d_alm, nb*nlm*sizeof(thrust::complex<double>));
  hipMalloc(&d_almconj, nb*nlm*sizeof(thrust::complex<double>));

  hipMemcpy(d_alm, alm, nb*nlm*sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
  hipMemcpy(d_almconj, almconj, nb*nlm*sizeof(thrust::complex<double>), hipMemcpyHostToDevice);

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

  add_to_power5_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_fivepcf,
        d_weight5pcf, d_alm, d_almconj, lut5_l1, lut5_l2,
        lut5_l12, lut5_l3, lut5_l4, 
        lut5_n, lut5_zeta, lut5_i, lut5_j, lut5_k, lut5_l,
        wp, nb, norder, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  hipFree(d_alm);
  hipFree(d_almconj);
}

void gpu_add_to_power5_float(float *d_fivepcf, float *d_weight5pcf, Complex* alm,
        Complex *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l12, int *lut5_l3,
        int *lut5_l4, int *lut5_n,
        int *lut5_zeta, int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
        float wp, int nb, int norder, int nlm, int nouter, int ninner, int nell5) {

  thrust::complex<float>* d_alm, *d_almconj;
  hipMallocManaged(&d_alm, nb*nlm*sizeof(thrust::complex<float>));
  hipMallocManaged(&d_almconj, nb*nlm*sizeof(thrust::complex<float>));

  for (int i = 0; i < nb*nlm; i++) {
    d_alm[i] = (thrust::complex<float>)(alm[i]);
    d_almconj[i] = (thrust::complex<float>)(almconj[i]);
  }

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

  add_to_power5_kernel_float<<<blocksPerGrid, threadsPerBlock>>>(d_fivepcf,
	d_weight5pcf, d_alm, d_almconj, lut5_l1, lut5_l2, lut5_l12,
	lut5_l3, lut5_l4, lut5_n, lut5_zeta, lut5_i,
	lut5_j, lut5_k, lut5_l, wp, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  hipFree(d_alm);
  hipFree(d_almconj);
}

void gpu_add_to_power5_mixed(double *d_fivepcf, double *d_weight5pcf, Complex* alm,
        Complex *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l12, int *lut5_l3,
        int *lut5_l4, int *lut5_n,
        int *lut5_zeta, int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
        float wp, int nb, int norder, int nlm, int nouter, int ninner, int nell5) {

  thrust::complex<float>* d_alm, *d_almconj;
  hipMallocManaged(&d_alm, nb*nlm*sizeof(thrust::complex<float>));
  hipMallocManaged(&d_almconj, nb*nlm*sizeof(thrust::complex<float>));

  for (int i = 0; i < nb*nlm; i++) {
    d_alm[i] = (thrust::complex<float>)(alm[i]);
    d_almconj[i] = (thrust::complex<float>)(almconj[i]);
  }

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

  add_to_power5_kernel_mixed<<<blocksPerGrid, threadsPerBlock>>>(d_fivepcf,
	d_weight5pcf, d_alm, d_almconj, lut5_l1, lut5_l2, lut5_l12,
	lut5_l3, lut5_l4, lut5_n, lut5_zeta, lut5_i, lut5_j,
	lut5_k, lut5_l, wp, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  hipFree(d_alm);
  hipFree(d_almconj);
}

void gpu_add_to_power5_with_memcpy(double *fivepcf, double *weight5pcf, Complex* alm,
        Complex *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l12, int *lut5_l3,
        int *lut5_l4, int *lut5_n,
        int *lut5_zeta, int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
	double wp, int nb, int norder, int nlm, int nouter, int ninner, int nell5) {


  double* d_fivepcf, *d_weight5pcf;
  thrust::complex<double>* d_alm, *d_almconj;
  size_t size_w = sizeof(double)*(norder+1)*(norder+1)*(norder+1)*(norder+1)*(2*norder+1)*(norder+1)*(norder+1)*(norder+1);
  size_t size_5 = sizeof(double)*nell5*ninner;

  hipMalloc(&d_fivepcf, size_5); 
  hipMalloc(&d_weight5pcf, size_w); 
  hipMalloc(&d_alm, nb*nlm*sizeof(thrust::complex<double>));
  hipMalloc(&d_almconj, nb*nlm*sizeof(thrust::complex<double>));

  hipMemcpy(d_alm, alm, nb*nlm*sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
  hipMemcpy(d_almconj, almconj, nb*nlm*sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
  hipMemcpy(d_fivepcf, fivepcf, size_5, hipMemcpyHostToDevice);
  hipMemcpy(d_weight5pcf, weight5pcf, size_w, hipMemcpyHostToDevice);

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

  add_to_power5_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_fivepcf,
        d_weight5pcf, d_alm, d_almconj, lut5_l1, lut5_l2,
        lut5_l12, lut5_l3, lut5_l4, 
        lut5_n, lut5_zeta, lut5_i, lut5_j, lut5_k, lut5_l,
        wp, nb, norder, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
  //Copy memory back to host in this method
  hipMemcpy(fivepcf, d_fivepcf, size_5, hipMemcpyDeviceToHost);
  //have to free memory
  hipFree(d_alm);
  hipFree(d_almconj);
  hipFree(d_fivepcf);
  hipFree(d_weight5pcf);
}

void gpu_add_to_power5_orig(double *d_fivepcf, double *d_weight5pcf, Complex* alm,
        Complex *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l3,
        int *lut5_l4, int *lut5_m1, int *lut5_m2, int *lut5_m3, int *lut5_n,
        int *lut5_zeta, int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
	double wp, int nb, int norder, int nlm, int nouter, int ninner, int nell5) {

  thrust::complex<double>* d_alm, *d_almconj;

  hipMalloc(&d_alm, nb*nlm*sizeof(thrust::complex<double>));
  hipMalloc(&d_almconj, nb*nlm*sizeof(thrust::complex<double>));

  hipMemcpy(d_alm, alm, nb*nlm*sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
  hipMemcpy(d_almconj, almconj, nb*nlm*sizeof(thrust::complex<double>), hipMemcpyHostToDevice);

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

  add_to_power5_kernel_orig<<<blocksPerGrid, threadsPerBlock>>>(d_fivepcf,
        d_weight5pcf, d_alm, d_almconj, lut5_l1,lut5_l2,
        lut5_l3, lut5_l4, lut5_m1, lut5_m2, lut5_m3,
        lut5_n, lut5_zeta, lut5_i, lut5_j, lut5_k, lut5_l,
        wp, nb, norder, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  hipFree(d_alm);
  hipFree(d_almconj);
}

void gpu_add_to_power5_orig_float(float *d_fivepcf, float *d_weight5pcf, Complex* alm,
        Complex *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l3,
        int *lut5_l4, int *lut5_m1, int *lut5_m2, int *lut5_m3, int *lut5_n,
        int *lut5_zeta, int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
        float wp, int nb, int norder, int nlm, int nouter, int ninner, int nell5) {

  thrust::complex<float>* d_alm, *d_almconj;
  hipMallocManaged(&d_alm, nb*nlm*sizeof(thrust::complex<float>));
  hipMallocManaged(&d_almconj, nb*nlm*sizeof(thrust::complex<float>));

  for (int i = 0; i < nb*nlm; i++) {
    d_alm[i] = (thrust::complex<float>)(alm[i]);
    d_almconj[i] = (thrust::complex<float>)(almconj[i]);
  }

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

  add_to_power5_kernel_orig_float<<<blocksPerGrid, threadsPerBlock>>>(d_fivepcf,
        d_weight5pcf, d_alm, d_almconj, lut5_l1,lut5_l2,
        lut5_l3, lut5_l4, lut5_m1, lut5_m2, lut5_m3,
        lut5_n, lut5_zeta, lut5_i, lut5_j, lut5_k, lut5_l,
        wp, nb, norder, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  hipFree(d_alm);
  hipFree(d_almconj);
}

void gpu_add_to_power5_orig_mixed(double *d_fivepcf, double *d_weight5pcf, Complex* alm,
        Complex *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l3,
        int *lut5_l4, int *lut5_m1, int *lut5_m2, int *lut5_m3, int *lut5_n,
        int *lut5_zeta, int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
        float wp, int nb, int norder, int nlm, int nouter, int ninner, int nell5) {

  thrust::complex<float>* d_alm, *d_almconj;
  hipMallocManaged(&d_alm, nb*nlm*sizeof(thrust::complex<float>));
  hipMallocManaged(&d_almconj, nb*nlm*sizeof(thrust::complex<float>));

  for (int i = 0; i < nb*nlm; i++) {
    d_alm[i] = (thrust::complex<float>)(alm[i]);
    d_almconj[i] = (thrust::complex<float>)(almconj[i]);
  }

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

  add_to_power5_kernel_orig_mixed<<<blocksPerGrid, threadsPerBlock>>>(d_fivepcf,
        d_weight5pcf, d_alm, d_almconj, lut5_l1,lut5_l2,
        lut5_l3, lut5_l4, lut5_m1, lut5_m2, lut5_m3,
        lut5_n, lut5_zeta, lut5_i, lut5_j, lut5_k, lut5_l,
        wp, nb, norder, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  hipFree(d_alm);
  hipFree(d_almconj);
}

void gpu_add_to_power5_orig_with_memcpy(double *fivepcf, double *weight5pcf, Complex* alm,
        Complex *almconj, int *lut5_l1, int *lut5_l2, int *lut5_l3,
        int *lut5_l4, int *lut5_m1, int *lut5_m2, int *lut5_m3, int *lut5_n,
        int *lut5_zeta, int *lut5_i, int *lut5_j, int *lut5_k, int *lut5_l,
	double wp, int nb, int norder, int nlm, int nouter, int ninner, int nell5) {

  double* d_fivepcf, *d_weight5pcf;
  thrust::complex<double>* d_alm, *d_almconj;
  size_t size_w = sizeof(double)*(norder+1)*(norder+1)*(norder+1)*(norder+1)*(2*norder+1)*(norder+1)*(norder+1)*(norder+1);
  size_t size_5 = sizeof(double)*nell5*ninner;

  hipMalloc(&d_fivepcf, size_5); 
  hipMalloc(&d_weight5pcf, size_w); 
  hipMalloc(&d_alm, nb*nlm*sizeof(thrust::complex<double>));
  hipMalloc(&d_almconj, nb*nlm*sizeof(thrust::complex<double>));

  hipMemcpy(d_fivepcf, fivepcf, size_5, hipMemcpyHostToDevice);
  hipMemcpy(d_weight5pcf, weight5pcf, size_w, hipMemcpyHostToDevice);
  hipMemcpy(d_alm, alm, nb*nlm*sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
  hipMemcpy(d_almconj, almconj, nb*nlm*sizeof(thrust::complex<double>), hipMemcpyHostToDevice);

  // Invoke kernel
  int threadsPerBlock = 512;
  long threads = ninner*nouter;
  int blocksPerGrid = (threads+threadsPerBlock-1) / threadsPerBlock;

  add_to_power5_kernel_orig<<<blocksPerGrid, threadsPerBlock>>>(d_fivepcf,
        d_weight5pcf, d_alm, d_almconj, lut5_l1,lut5_l2,
        lut5_l3, lut5_l4, lut5_m1, lut5_m2, lut5_m3,
        lut5_n, lut5_zeta, lut5_i, lut5_j, lut5_k, lut5_l,
        wp, nb, norder, nlm, nouter, ninner);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  hipMemcpy(fivepcf, d_fivepcf, size_5, hipMemcpyDeviceToHost);
  hipFree(d_alm);
  hipFree(d_almconj);
  hipFree(d_fivepcf);
  hipFree(d_weight5pcf);
}
